#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>

// CUDA Headers
#include "hip/hip_runtime.h"



// Helper definition
#define VAR(v, i) __typeof(i) v=(i)
#define FOR(i, j, k) for (int i = (j); i <= (k); ++i)
#define FORD(i, j, k)for (int i=(j); i >= (k); --i)
#define FORE(i, c) for(VAR(i, (c).begin()); i != (c).end(); ++i)
#define REP(i, n) for(int i = 0;i <(n); ++i)

// CONSTS
#define INF 	1061109567 // 3F 3F 3F 3F
#define CHARINF 63	   // 3F	
#define CHARBIT 8
#define NONE	-1

#define CMCPYHTD hipMemcpyHostToDevice
#define CMCPYDTH hipMemcpyDeviceToHost

// CONSTS for compute capability 2.0
#define BLOCK_WIDTH 16
#define WARP 	    32

bool gPrint = false; 	// print graph d or not
bool gDebug = false;	// print more deatails to debug

/** Cuda handle error, if err is not success print error and line in code
*
* @param status CUDA Error types
*/
#define HANDLE_ERROR(err) \
{ \
	if (err != hipSuccess) \
	{ \
		fprintf(stderr, "%s failed  at line %d \nError message: %s \n", \
			__FILE__, __LINE__ ,hipGetErrorString(err)); \
		exit(EXIT_FAILURE); \
	} \
}

/**Kernel for wake gpu
*
* @param reps dummy variable only to perform some action
*/
__global__ void wake_gpu_kernel(int reps) 
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= reps) return;
}

/**Kernel for parallel Floyd Warshall algorithm on gpu
* 
* @param u number vertex of which is performed relaxation paths [v1, v2]
* @param n number of vertices in the graph G:=(V,E), n := |V(G)|
* @param d matrix of shortest paths d(G)
* @param p matrix of predecessors p(G)
*/
__global__ void fw_kernel(const unsigned int u, const unsigned int n, int * const d, int * const p)
{
	int v1 = blockDim.y * blockIdx.y + threadIdx.y;
	int v2 = blockDim.x * blockIdx.x + threadIdx.x;
	
	if (v1 < n && v2 < n) 
	{
		int newPath = d[v1 * n + u] + d[u * n + v2];
		int oldPath = d[v1 * n + v2];
		if (oldPath > newPath)
		{
			d[v1 * n + v2] = newPath;
			p[v1 * n + v2] = p[u * n + v2];		
		}
	}
}

/** Parallel Floyd Warshall algorithm using gpu
*
* @param n number of vertices in the graph G:=(V,E), n := |V(G)|
* @param G is a the graph G:=(V,E)
* @param d matrix of shortest paths d(G)
* @param p matrix of predecessors p(G)
*/
extern "C" void fw_gpu(const unsigned int n, const int * const G, int * const d, int * const p)
{
	int *dev_d = 0;
	int *dev_p = 0;
	hipError_t cudaStatus;
	hipStream_t cpyStream;

	// Choose which GPU to run on, change this on a multi-GPU system.
    	cudaStatus = hipSetDevice(0);
	HANDLE_ERROR(cudaStatus);

	// Initialize the grid and block dimensions here
	dim3 dimGrid((n - 1) / BLOCK_WIDTH + 1, (n - 1) / BLOCK_WIDTH + 1, 1); 
	dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH, 1);

	if (gDebug) 
	{
		printf("|V| %d\n", n);
		printf("Dim Grid:\nx - %d\ny - %d\nz - %d\n", dimGrid.x, dimGrid.y, dimGrid.z);
		printf("Dim Block::\nx - %d\ny - %d\nz - %d\n", dimBlock.x, dimBlock.y, dimBlock.z);
	}

	// Create new stream to copy data	
	cudaStatus = hipStreamCreate(&cpyStream);
	HANDLE_ERROR(cudaStatus);

	// Allocate GPU buffers for matrix of shortest paths d(G) and predecessors p(G)
	cudaStatus =  hipMalloc((void**)&dev_d, n * n * sizeof(int));
	HANDLE_ERROR(cudaStatus);
	cudaStatus =  hipMalloc((void**)&dev_p, n * n * sizeof(int));
	HANDLE_ERROR(cudaStatus);
	
	// Wake up gpu
	wake_gpu_kernel<<<1, dimBlock>>>(32);

        // Copy input from host memory to GPU buffers.
        cudaStatus = hipMemcpyAsync(dev_d, G, n * n * sizeof(int), CMCPYHTD, cpyStream);
	cudaStatus = hipMemcpyAsync(dev_p, p, n * n * sizeof(int), CMCPYHTD, cpyStream);

        // cudaDeviceSynchronize waits for the kernel to finish, and returns
        cudaStatus = hipDeviceSynchronize();
        HANDLE_ERROR(cudaStatus);

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(fw_kernel), hipFuncCachePreferL1 );
	FOR(u, 0, n - 1) 
	{
		fw_kernel<<<dimGrid, dimBlock>>>(u, n, dev_d, dev_p);
	}

	// Check for any errors launching the kernel
    	cudaStatus = hipGetLastError();
	HANDLE_ERROR(cudaStatus);

        // cudaDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	HANDLE_ERROR(cudaStatus);
	
	cudaStatus = hipMemcpy(d, dev_d, n * n * sizeof(int), CMCPYDTH);
	HANDLE_ERROR(cudaStatus);

	cudaStatus = hipMemcpy(p, dev_p, n * n * sizeof(int), CMCPYDTH);
	HANDLE_ERROR(cudaStatus);

	cudaStatus = hipFree(dev_d);
	HANDLE_ERROR(cudaStatus);

	cudaStatus = hipFree(dev_p);
	HANDLE_ERROR(cudaStatus);

	return;
}

/**
* Print graph G as a matrix
*
* @param n number of vertices in the graph G:=(V,E), n := |V(G)|
* @param G is a the graph G:=(V,E)
*/
void print_graph(const unsigned int n, const int * const G)
{
	FOR(v1, 0, n - 1)
	{
		FOR(v2, 0, n - 1) 
		{	
			if (G[v1 * n + v2] < INF)
				printf("%d ", G[v1 * n + v2]);
			else
				printf("INF ");
		}
		printf("\n");
	}
	printf("\n");
}

/**
* Reconstruct Path
*
* @param i, j id vertex 
* @param G is a the graph G:=(V,E)
* @param p matrix of predecessors p(G)
*/
int reconstruct_path(unsigned int n, unsigned int i, unsigned int j, const int * const p, const int * const G)
{
	if (i == j )
		return 0;
	else if ( p[i * n + j] == NONE)
		return INF;
	else
	{
		int path = reconstruct_path(n, i, p[i * n + j], p, G);
		if (path == INF) 
			return INF;
		else
			return path + G[ p [i * n + j] * n + j];
	}
}

/**
* Check paths
*
* @param n number of vertices in the graph G:=(V,E), n := |V(G)|
* @param G is a the graph G:=(V,E)
* @param d matrix of shortest paths d(G)
* @param p matrix of predecessors p(G)
*/
bool check_paths(const unsigned int n, const int * const G, const int * const d, const int * const p)
{
	
	FOR (i, 0, n - 1)
	{
		FOR (j, 0, n - 1)
		{
			int path = reconstruct_path(n, i, j, p, G);
			if (gDebug)
				printf("%d %d %d == %d \n", i, j, path, d[i * n + j]);
			if (path != d[i * n + j])
				return false;
		}
	}

	return true;
}
